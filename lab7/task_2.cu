#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
     unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

     if(i < N) {
        c[i] = a[i] + b[i];
     }
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays
    host_array_a = (float *) malloc(N * sizeof(float));
    host_array_b = (float *) malloc(N * sizeof(float));
    host_array_c = (float *) malloc(N * sizeof(float));

    // TODO 2: Allocate the device's arrays
    hipMalloc((void **)&device_array_a, N * sizeof(float));
    hipMalloc((void **)&device_array_b, N * sizeof(float));
    hipMalloc((void **)&device_array_c, N * sizeof(float));

    // TODO 3: Check for allocation errors
    if(!host_array_a | !host_array_b | !host_array_c | 
        !device_array_a | !device_array_b | !device_array_c) {
        fprintf(stderr, "Eroare malloc\n");
        exit(-1);
    }

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.
    fill_array_float(host_array_a, N);
    fill_array_random(host_array_b, N);

    // TODO 5: Copy the host's arrays to device
    hipMemcpy(device_array_a, host_array_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, N * sizeof(float), hipMemcpyHostToDevice);

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).
    int block_size = 256;
    int block_count = N / block_size;
    if(N % 256) {
        block_count++;
    }

    add_arrays<<<block_count, block_size>>>(device_array_a, device_array_b, device_array_c, N);
    hipDeviceSynchronize();

    // TODO 7: Copy back the results and then uncomment the checking function
    hipMemcpy(host_array_c, device_array_c, N * sizeof(float), hipMemcpyDeviceToHost);

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory
    free(host_array_a);
    free(host_array_b);
    free(host_array_c);

    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);
   
    return 0;
}